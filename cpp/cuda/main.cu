
#include <hip/hip_runtime.h>
#include <vector>

// __global__ 키워드를 붙이면 device에서 작동된다.
__global__ void add(int n, float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int N = 300000000;
    std::vector<float> a(N, 1.0f);
    std::vector<float> b(N, 1.0f);
    std::vector<float> c(N, 1.0f);

    float *dev_a, *dev_b, *dev_c;
    hipMalloc(&dev_a, N * sizeof(float));
    hipMalloc(&dev_b, N * sizeof(float));
    hipMalloc(&dev_c, N * sizeof(float));

    hipMemcpy(dev_a, a.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), N * sizeof(float), hipMemcpyHostToDevice);

    add<<<(N + 255) / 256, 256>>>(N, dev_a, dev_b, dev_c);

    hipMemcpy(c.data(), dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}